#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"

template <class T>
__global__ void parallel_transpose(T *out, T const *in, int nx, int ny) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= nx || y >= ny) return;
    out[y * nx + x] = in[x * nx + y];
}

int main() {
    int nx = 1<<14, ny = 1<<14;
    std::vector<int, CudaAllocator<int>> in(nx * ny);
    std::vector<int, CudaAllocator<int>> out(nx * ny);

    for (int i = 0; i < nx * ny; i++) {
        in[i] = i;
    }

    TICK(parallel_transpose);
    parallel_transpose<<<dim3(nx / 32, ny / 32, 1), dim3(32, 32, 1)>>>
        (out.data(), in.data(), nx, ny);
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(parallel_transpose);

    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            if (out[y * nx + x] != in[x * nx + y]) {
                printf("Wrong At x=%d,y=%d: %d != %d\n", x, y,
                       out[y * nx + x], in[x * nx + y]);
                return -1;
            }
        }
    }

    printf("All Correct!\n");
    return 0;
}
