#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include <cmath>
#include "ticktock.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "freshman.h"




__global__ void kernel(int * arr, int n){
    for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n; i += blockDim.x * gridDim.x){
        arr[i] = i;
    }
}

int main(int argc,char **argv)
{
    int n = 65535;
    int * arr;
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));

    int threadsPerBlock = 128;
    int blocksPerGrid = (n + threadsPerBlock -1)/ threadsPerBlock;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(arr, n);

    checkCudaErrors(hipDeviceSynchronize());

    for(int i = 0; i < n ; i++){
        printf("arr[%d] : %d\n", i, arr[i]);
    }
    hipFree(arr);
    return 0;
}